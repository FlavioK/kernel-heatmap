#include "hip/hip_runtime.h"
#include "utility_func.cuh"
#include "utility_host.hpp"
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

// Prints a message and returns zero if the given value is not hipSuccess
#define CheckCUDAError(val) (InternalCheckCUDAError((val), #val, __FILE__, __LINE__))

// Called internally by CheckCUDAError
static inline int InternalCheckCUDAError(hipError_t result, const char *fn,
        const char *file, int line) {
    if (result == hipSuccess) return 0;
    printf("CUDA error %d in %s, line %d (%s): %s\n", (int) result, file, line,
            fn, hipGetErrorString(result));
    return -1;
}

static __global__ void getStartTimeInternal(uint64_t *targetStartTime) {
    if(threadIdx.x == 0){
        *targetStartTime = getTime() + START_TIME_OFFSET_NS;
    }
    __syncthreads();
}

void getStartTime(uint64_t *targetStartTime){
        getStartTimeInternal<<<1,1>>>(targetStartTime);

        if (CheckCUDAError(hipDeviceSynchronize())) perror("Could not synchronize device\n");
}
